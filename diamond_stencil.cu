#include "hip/hip_runtime.h"
//===--------------------------------------------------------------------------------*-
// C++ -*-===//
//                          _
//                         | |
//                       __| | __ ___      ___ ___
//                      / _` |/ _` \ \ /\ / / '_  |
//                     | (_| | (_| |\ V  V /| | | |
//                      \__,_|\__,_| \_/\_/ |_| |_| - Compiler Toolchain
//
//
//  This file is distributed under the MIT License (MIT).
//  See LICENSE.txt for details.
//
//===------------------------------------------------------------------------------------------===//

#include "diamond_stencil.h"

#include <atlas/mesh/Elements.h>
#include <atlas/mesh/HybridElements.h>
#include <atlas/mesh/Nodes.h>
#include <atlas/util/CoordinateEnums.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

namespace {

#define gpuErrchk(ans)                                                                             \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
  if(code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if(abort)
      exit(code);
  }
}

#define E_C_V_SIZE 4
#define BLOCK_SIZE 128
#define DEVICE_MISSING_VALUE -1

__global__ void compute_vn(int numEdges, int kSize, const int* __restrict__ ecvTable,
                           double* __restrict__ vn_vert, const double* __restrict__ u_vert,
                           const double* __restrict__ v_vert,
                           const double* __restrict__ primal_normal_vert_x,
                           const double* __restrict__ primal_normal_vert_y) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  {
    for(int kIter = 0; kIter < kSize; kIter++) {
      int offset = kIter * numEdges;
      for(int nbhIter = 0; nbhIter < E_C_V_SIZE; nbhIter++) { // for(e->c->v)
        int nbhIdx = __ldg(&ecvTable[pidx * E_C_V_SIZE + nbhIter]);
        if(nbhIdx == DEVICE_MISSING_VALUE) {
          continue;
        }
        vn_vert[offset + pidx * E_C_V_SIZE + nbhIter] =
            __ldg(&u_vert[offset + nbhIdx]) *
                __ldg(&primal_normal_vert_x[offset + pidx * E_C_V_SIZE + nbhIter]) +
            __ldg(&v_vert[offset + nbhIdx]) *
                __ldg(&primal_normal_vert_y[offset + pidx * E_C_V_SIZE + nbhIter]);
      }
    }
  }
}

__global__ void reduce_dvt_tang(int numEdges, const int* __restrict__ ecvTable,
                                double* __restrict__ dvt_tang, const double* __restrict__ u_vert,
                                const double* __restrict__ v_vert,
                                const double* __restrict__ dual_normal_vert_x,
                                const double* __restrict__ dual_normal_vert_y) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  double weights[E_C_V_SIZE] = {-1., 1., 0., 0.};
  {
    double lhs = 0.;
    for(int nbhIter = 0; nbhIter < E_C_V_SIZE; nbhIter++) { // for(e->c->v)
      int nbhIdx = __ldg(&ecvTable[pidx * E_C_V_SIZE + nbhIter]);
      if(nbhIdx == DEVICE_MISSING_VALUE) {
        continue;
      }
      lhs += weights[nbhIter] * __ldg(&u_vert[nbhIdx]) *
                 __ldg(&dual_normal_vert_x[pidx * E_C_V_SIZE + nbhIter]) +
             __ldg(&v_vert[nbhIdx]) * __ldg(&dual_normal_vert_y[pidx * E_C_V_SIZE + nbhIter]);
    }
    dvt_tang[pidx] = lhs;
  }
}

__global__ void finish_dvt_tang(int numEdges, double* __restrict__ dvt_tang,
                                const double* __restrict__ tangent_orientation) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  dvt_tang[pidx] = dvt_tang[pidx] * __ldg(&tangent_orientation[pidx]);
}

__global__ void reduce_dvt_norm(int numEdges, const int* __restrict__ ecvTable,
                                double* __restrict__ dvt_norm, const double* __restrict__ u_vert,
                                const double* __restrict__ v_vert,
                                const double* __restrict__ dual_normal_vert_x,
                                const double* __restrict__ dual_normal_vert_y) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  double weights[E_C_V_SIZE] = {0., 0., -1., 1.};
  {
    double lhs = 0.;
    for(int nbhIter = 0; nbhIter < E_C_V_SIZE; nbhIter++) { // for(e->c->v)
      int nbhIdx = __ldg(&ecvTable[pidx * E_C_V_SIZE + nbhIter]);
      if(nbhIdx == DEVICE_MISSING_VALUE) {
        continue;
      }
      lhs += weights[nbhIter] * __ldg(&u_vert[nbhIdx]) *
                 __ldg(&dual_normal_vert_x[pidx * E_C_V_SIZE + nbhIter]) +
             __ldg(&v_vert[nbhIdx]) * __ldg(&dual_normal_vert_y[pidx * E_C_V_SIZE + nbhIter]);
    }
    dvt_norm[pidx] = lhs;
  }
}

__global__ void smagorinsky_1(int numEdges, const int* __restrict__ ecvTable,
                              double* __restrict__ kh_smag_1, const double* __restrict__ vn_vert) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  double weights[E_C_V_SIZE] = {-1., 1., 0., 0.};
  {
    double lhs = 0.;
    for(int nbhIter = 0; nbhIter < E_C_V_SIZE; nbhIter++) { // for(e->c->v)
      int nbhIdx = __ldg(&ecvTable[pidx * E_C_V_SIZE + nbhIter]);
      if(nbhIdx == DEVICE_MISSING_VALUE) {
        continue;
      }
      lhs += vn_vert[nbhIdx] * weights[nbhIter];
    }
    kh_smag_1[pidx] = lhs;
  }
} // namespace

__global__ void smagorinsky_1_multitply_facs(int numEdges, double* __restrict__ kh_smag_1,
                                             const double* __restrict__ tangent_orientation,
                                             const double* __restrict__ inv_vert_vert_length,
                                             const double* __restrict__ inv_primal_edge_length,
                                             const double* __restrict__ dvt_norm) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  kh_smag_1[pidx] =
      kh_smag_1[pidx] * __ldg(&inv_primal_edge_length[pidx]) * __ldg(&tangent_orientation[pidx]) -
      __ldg(&dvt_norm[pidx]) * __ldg(&inv_vert_vert_length[pidx]);
}

__global__ void smagorinsky_1_square(int numEdges, double* __restrict__ kh_smag_1) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  kh_smag_1[pidx] = kh_smag_1[pidx] * kh_smag_1[pidx];
}

__global__ void smagorinsky_2(int numEdges, const int* __restrict__ ecvTable,
                              double* __restrict__ kh_smag_2, const double* __restrict__ vn_vert) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  double weights[E_C_V_SIZE] = {0., 0., -1., 1.};
  {
    double lhs = 0.;
    for(int nbhIter = 0; nbhIter < E_C_V_SIZE; nbhIter++) { // for(e->c->v)
      int nbhIdx = __ldg(&ecvTable[pidx * E_C_V_SIZE + nbhIter]);
      if(nbhIdx == DEVICE_MISSING_VALUE) {
        continue;
      }
      lhs += vn_vert[nbhIdx] * weights[nbhIter];
    }
    kh_smag_2[pidx] = lhs;
  }
}

__global__ void smagorinsky_2_multitply_facs(int numEdges, double* __restrict__ kh_smag_2,
                                             const double* __restrict__ inv_vert_vert_length,
                                             const double* __restrict__ inv_primal_edge_length,
                                             const double* __restrict__ dvt_tang) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  kh_smag_2[pidx] = kh_smag_2[pidx] * __ldg(&inv_vert_vert_length[pidx]) -
                    __ldg(&dvt_tang[pidx]) * __ldg(&inv_primal_edge_length[pidx]);
}

__global__ void smagorinsky_2_square(int numEdges, double* __restrict__ kh_smag_2) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  kh_smag_2[pidx] = kh_smag_2[pidx] * kh_smag_2[pidx];
}

__global__ void smagorinsky(int numEdges, double* __restrict__ kh_smag,
                            const double* __restrict__ kh_smag_1,
                            const double* __restrict__ kh_smag_2) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  kh_smag[pidx] = sqrt(kh_smag_1[pidx] + kh_smag_2[pidx]);
}

__global__ void diamond(int numEdges, int kSize, const int* __restrict__ ecvTable,
                        double* __restrict__ nabla2, const double* __restrict__ vn_vert,
                        const double* __restrict__ inv_primal_edge_length,
                        const double* __restrict__ inv_vert_vert_length) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  double weights[E_C_V_SIZE] = {
      __ldg(&inv_primal_edge_length[pidx]) * __ldg(&inv_primal_edge_length[pidx]),
      __ldg(&inv_primal_edge_length[pidx]) * __ldg(&inv_primal_edge_length[pidx]),
      __ldg(&inv_vert_vert_length[pidx]) * __ldg(&inv_vert_vert_length[pidx]),
      __ldg(&inv_vert_vert_length[pidx]) * __ldg(&inv_vert_vert_length[pidx])};

  for(int kIter = 0; kIter < kSize; kIter++) {
    int offset = kIter * numEdges;
    double lhs = 0.;
    for(int nbhIter = 0; nbhIter < E_C_V_SIZE; nbhIter++) { // for(e->c->v)
      int nbhIdx = __ldg(&ecvTable[offset + pidx * E_C_V_SIZE + nbhIter]);
      if(nbhIdx == DEVICE_MISSING_VALUE) {
        continue;
      }
      lhs += 4. * vn_vert[offset + pidx * E_C_V_SIZE + nbhIter] * weights[offset + nbhIter];
    }
    nabla2[pidx] = lhs;
  }
}

__global__ void nabla2(int numEdges, int kSize, double* __restrict__ nabla2,
                       double* __restrict__ vn, const double* __restrict__ inv_primal_edge_length,
                       const double* __restrict__ inv_vert_vert_length) {
  unsigned int pidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(pidx >= numEdges) {
    return;
  }
  for(int kIter = 0; kIter < kSize; kIter++) {
    int offset = kIter * numEdges;
    nabla2[offset + pidx] =
        nabla2[offset + pidx] -
        8. * __ldg(&vn[offset + pidx]) * __ldg(&inv_primal_edge_length[offset + pidx]) *
            __ldg(&inv_primal_edge_length[offset + pidx]) -
        8. * __ldg(&vn[offset + pidx]) * __ldg(&inv_vert_vert_length[offset + pidx]) *
            __ldg(&inv_vert_vert_length[offset + pidx]);
  }
}

} // namespace

void generateNbhTable(atlas::Mesh const& mesh, std::vector<dawn::LocationType> chain,
                      int numElements, int numNbhPerElement, int* target) {
  std::vector<atlas::idx_t> elems;
  switch(chain.front()) {
  case dawn::LocationType::Cells: {
    for(auto cell : atlasInterface::getCells(atlasInterface::atlasTag(), mesh)) {
      elems.push_back(cell);
    }
    break;
  }
  case dawn::LocationType::Edges: {
    for(auto edge : atlasInterface::getEdges(atlasInterface::atlasTag(), mesh)) {
      elems.push_back(edge);
    }
    break;
  }
  case dawn::LocationType::Vertices: {
    for(auto vertex : atlasInterface::getVertices(atlasInterface::atlasTag(), mesh)) {
      elems.push_back(vertex);
    }
    break;
  }
  }

  assert(elems.size() == numElements);

  std::vector<int> hostTable;
  for(int elem : elems) {
    auto neighbors = atlasInterface::getNeighbors(mesh, chain, elem);
    for(int nbhIdx = 0; nbhIdx < numNbhPerElement; nbhIdx++) {
      if(nbhIdx < neighbors.size()) {
        hostTable.push_back(neighbors[nbhIdx]);
      } else {
        hostTable.push_back(DEVICE_MISSING_VALUE);
      }
    }
  }

  assert(hostTable.size() == numElements * numNbhPerElement);
  gpuErrchk(hipMemcpy(target, hostTable.data(), sizeof(int) * numElements * numNbhPerElement,
                       hipMemcpyHostToDevice));
}

GpuTriMesh::GpuTriMesh(const atlas::Mesh& mesh) {

  numNodes_ = mesh.nodes().size();
  numEdges_ = mesh.edges().size();
  numCells_ = mesh.cells().size();

  // position vector
  gpuErrchk(hipMalloc((void**)&pos_, sizeof(double2) * mesh.nodes().size()));

  gpuErrchk(hipMalloc((void**)&ecvTable_, sizeof(int) * mesh.edges().size() * E_C_V_SIZE));
  // copy position vector
  std::vector<double2> pHost;
  auto xy = atlas::array::make_view<double, 2>(mesh.nodes().xy());
  for(int nodeIdx = 0; nodeIdx < mesh.nodes().size(); nodeIdx++) {
    pHost.push_back({xy(nodeIdx, atlas::LON), xy(nodeIdx, atlas::LAT)});
  }
  gpuErrchk(hipMemcpy(pos_, pHost.data(), sizeof(double2) * mesh.nodes().size(),
                       hipMemcpyHostToDevice));
  generateNbhTable(
      mesh, {dawn::LocationType::Edges, dawn::LocationType::Cells, dawn::LocationType::Vertices},
      mesh.edges().size(), E_C_V_SIZE, ecvTable_);
}

#define initField(field, cudaStorage)                                                              \
  {                                                                                                \
    gpuErrchk(hipMalloc((void**)&cudaStorage, sizeof(double) * field.numElements()));             \
    gpuErrchk(hipMemcpy(cudaStorage, field.data(), sizeof(double) * field.numElements(),          \
                         hipMemcpyHostToDevice));                                                 \
  }

DiamondStencil::diamond_stencil::diamond_stencil(
    const atlas::Mesh& mesh, int k_size, const atlasInterface::Field<double>& diff_multfac_smag,
    const atlasInterface::Field<double>& tangent_orientation,
    const atlasInterface::Field<double>& inv_primal_edge_length,
    const atlasInterface::Field<double>& inv_vert_vert_length,
    const atlasInterface::Field<double>& u_vert, const atlasInterface::Field<double>& v_vert,
    const atlasInterface::SparseDimension<double>& primal_normal_vert_x,
    const atlasInterface::SparseDimension<double>& primal_normal_vert_y,
    const atlasInterface::SparseDimension<double>& dual_normal_vert_x,
    const atlasInterface::SparseDimension<double>& dual_normal_vert_y,
    const atlasInterface::SparseDimension<double>& vn_vert, const atlasInterface::Field<double>& vn,
    const atlasInterface::Field<double>& dvt_tang, const atlasInterface::Field<double>& dvt_norm,
    const atlasInterface::Field<double>& kh_smag_1, const atlasInterface::Field<double>& kh_smag_2,
    const atlasInterface::Field<double>& kh_smag_e, const atlasInterface::Field<double>& z_nabla2_e)
    : sbase("diamond_stencil"), mesh_(mesh), kSize_(k_size) {

  initField(diff_multfac_smag, diff_multfac_smag_);
  initField(tangent_orientation, tangent_orientation_);
  initField(inv_primal_edge_length, inv_primal_edge_length_);
  initField(inv_vert_vert_length, inv_vert_vert_length_);
  initField(u_vert, u_vert_);
  initField(v_vert, v_vert_);
  initField(primal_normal_vert_x, primal_normal_vert_x_);
  initField(primal_normal_vert_y, primal_normal_vert_y_);
  initField(dual_normal_vert_x, dual_normal_vert_x_);
  initField(dual_normal_vert_y, dual_normal_vert_y_);
  initField(vn_vert, vn_vert_);
  initField(vn, vn_);
  initField(dvt_tang, dvt_tang_);
  initField(dvt_norm, dvt_norm_);
  initField(kh_smag_1, kh_smag_1_);
  initField(kh_smag_2, kh_smag_2_);
  initField(kh_smag_e, kh_smag_e_);
  initField(z_nabla2_e, z_nabla2_e_);
}

void DiamondStencil::diamond_stencil::run() {
  // starting timers
  start();

  // stage over edges
  {
    dim3 dG((mesh_.NumEdges() + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 dB(BLOCK_SIZE);

    compute_vn<<<dG, dB>>>(mesh_.NumEdges(), kSize_, mesh_.ECVTable(), vn_vert_, u_vert_, v_vert_,
                           primal_normal_vert_x_, primal_normal_vert_y_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    reduce_dvt_tang<<<dG, dB>>>(mesh_.NumEdges(), mesh_.ECVTable(), dvt_tang_, u_vert_, v_vert_,
                                dual_normal_vert_x_, dual_normal_vert_y_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    finish_dvt_tang<<<dG, dB>>>(mesh_.NumEdges(), dvt_tang_, tangent_orientation_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    reduce_dvt_norm<<<dG, dB>>>(mesh_.NumEdges(), mesh_.ECVTable(), dvt_norm_, u_vert_, v_vert_,
                                dual_normal_vert_x_, dual_normal_vert_y_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    smagorinsky_1<<<dG, dB>>>(mesh_.NumEdges(), mesh_.ECVTable(), kh_smag_1_, vn_vert_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    smagorinsky_1_multitply_facs<<<dG, dB>>>(mesh_.NumEdges(), kh_smag_1_, tangent_orientation_,
                                             inv_vert_vert_length_, inv_primal_edge_length_,
                                             dvt_norm_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    smagorinsky_1_square<<<dG, dB>>>(mesh_.NumEdges(), kh_smag_1_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    smagorinsky_2<<<dG, dB>>>(mesh_.NumEdges(), mesh_.ECVTable(), kh_smag_2_, vn_vert_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    smagorinsky_2_multitply_facs<<<dG, dB>>>(mesh_.NumEdges(), kh_smag_2_, inv_vert_vert_length_,
                                             inv_primal_edge_length_, dvt_norm_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    smagorinsky_2_square<<<dG, dB>>>(mesh_.NumEdges(), kh_smag_2_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    smagorinsky<<<dG, dB>>>(mesh_.NumEdges(), kh_smag_e_, kh_smag_1_, kh_smag_2_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    diamond<<<dG, dB>>>(mesh_.NumEdges(), kSize_, mesh_.ECVTable(), z_nabla2_e_, vn_vert_,
                        inv_primal_edge_length_, inv_vert_vert_length_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    nabla2<<<dG, dB>>>(mesh_.NumEdges(), kSize_, z_nabla2_e_, vn_, inv_primal_edge_length_,
                       inv_vert_vert_length_);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
  }

  // stopping timers
  pause();
}

void DiamondStencil::diamond_stencil::CopyResultToHost(
    atlasInterface::Field<double>& kh_smag_e, atlasInterface::Field<double>& z_nabla2_e) const {
  gpuErrchk(hipMemcpy((double*)kh_smag_e.data(), kh_smag_e_,
                       sizeof(double) * kh_smag_e.numElements(), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy((double*)z_nabla2_e.data(), z_nabla2_e_,
                       sizeof(double) * z_nabla2_e.numElements(), hipMemcpyDeviceToHost));
}
